#include "hip/hip_runtime.h"
/* CUDA toy
 * Thong Nguyen, 2020 */

#include <cstdio>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "toygenerator.cuh"

__global__
void generate_goodness_of_fit_toys(float * dev_bkg_expected, 
                                   float * dev_obs_data,
                                   float * dev_q_toys,
                                   int n_bins,
                                   int ntoys,
                                   hiprandState *states,
                                   int trialsPerThread)
{
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int toy;
    float sum_log_likelihood;
    hiprand_init(42, tid, 0, &states[tid]); // Initialize CURAND
    for (int trial = 0; trial < trialsPerThread; trial++)
    {
        if ((trial+1) * tid < ntoys)
            {
            sum_log_likelihood = 0;
            for (int bin = 0; bin < n_bins; bin++)
            {
                toy = hiprand_poisson(&states[tid], dev_bkg_expected[bin]);
                sum_log_likelihood += chisquare(dev_bkg_expected[bin], toy);
            }
            dev_q_toys[tid * (trial+1)] = sum_log_likelihood;
        }
    }
}

__global__
void generate_neyman_pearson_toys(float * dev_bkg_expected, 
                                   float * dev_sig_expected,
                                   float * dev_obs_data,
                                   float * dev_q_toys,
                                   int n_bins,
                                   int ntoys,
                                   hiprandState *states,
                                   int trialsPerThread)
{
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int toy;
    float sum_log_likelihood, numerator, denominator;
    hiprand_init(42, tid, 0, &states[tid]); // Initialize CURAND
    for (int trial = 0; trial < trialsPerThread; trial++)
    {
        if ((trial+1) * tid < ntoys)
        {
            sum_log_likelihood = 0;
            for (int bin = 0; bin < n_bins; bin++)
            {
                toy = hiprand_poisson(&states[tid], dev_bkg_expected[bin]);
                numerator = log_poisson(dev_bkg_expected[bin]+dev_sig_expected[bin], toy);
                denominator = log_poisson(dev_bkg_expected[bin], toy);
                sum_log_likelihood += -2 * numerator/denominator;
            }
            dev_q_toys[tid * (trial+1)] = sum_log_likelihood;
        }
    }
}

void cuda_call_generate_goodness_of_fit_toys(int nBlocks,
                                            int threadsPerBlock,
                                            float * dev_bkg_expected, 
                                            float * dev_obs_data,
                                            float * dev_q_toys,
                                            int n_bins,
                                            int ntoys,
                                            hiprandState * devStates,
                                            int trialsPerThread)
{
    generate_goodness_of_fit_toys<<<nBlocks, threadsPerBlock>>>(dev_bkg_expected, 
                                                               dev_obs_data,
                                                               dev_q_toys,
                                                               n_bins,
                                                               ntoys,
                                                               devStates,
                                                               trialsPerThread);

}
    
void cuda_call_generate_neyman_pearson_toys(int nBlocks,
                                           int threadsPerBlock,
                                           float * dev_bkg_expected, 
                                           float * dev_sig_expected,
                                           float * dev_obs_data,
                                           float * dev_q_toys,
                                           int n_bins,
                                           int ntoys,
                                           hiprandState * devStates,
                                           int trialsPerThread)
{
    generate_neyman_pearson_toys<<<nBlocks, threadsPerBlock>>>(dev_bkg_expected, 
                                                               dev_sig_expected,
                                                               dev_obs_data,
                                                               dev_q_toys,
                                                               n_bins,
                                                               ntoys,
                                                               devStates,
                                                               trialsPerThread);
}
    

